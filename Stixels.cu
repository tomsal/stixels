#include "hip/hip_runtime.h"
/**
    This file is part of stixels. (https://github.com/dhernandez0/stixels).

    Copyright (c) 2016 Daniel Hernandez Juarez.

    stixels is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    stixels is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with stixels.  If not, see <http://www.gnu.org/licenses/>.

**/

#include "Stixels.hpp"

Stixels::Stixels() {
}

Stixels::~Stixels() {
}


void Stixels::Initialize() {
	m_disp_im_modified = m_disp_im;
	m_realcols = (m_cols-m_width_margin)/m_column_step;

	m_cost_table = new float[3*m_realcols];
	m_index_table = new int16_t[m_rows*3*m_realcols];

	m_max_sections = 50;

	m_stixels = new Section[m_realcols*m_max_sections];
	CUDA_CHECK_RETURN(hipMalloc((void **)&d_stixels, m_realcols*m_max_sections*sizeof(Section)));

	if(m_column_step > 1) {
		CUDA_CHECK_RETURN(hipHostMalloc((void**)&m_disp_im_modified, m_rows*m_realcols*sizeof(float)));
	}

	// Mean precomputation
	m_column = new pixel_t[m_rows*m_realcols];
	m_sum = new pixel_t[(m_rows+1)*m_realcols];
	m_valid = new pixel_t[(m_rows+1)*m_realcols];

	m_ground_function = new float[m_rows];

	// Log LUT, range: 0.0f - 1.0f
    m_log_lut = new float[LOG_LUT_SIZE+1];
	for(int i = 0; i < LOG_LUT_SIZE; i++) {
		const float log_res = (float)i/((float)LOG_LUT_SIZE);
		m_log_lut[i] = logf (log_res);
	}
    m_log_lut[LOG_LUT_SIZE] = 0.0f;

	// Frequently used values
	m_max_dis_log = logf(m_max_disf);
	m_rows_log = logf((float)m_rows);
	m_puniform_sky = m_max_dis_log - logf(m_pout_sky);
	m_puniform = m_max_dis_log - logf(m_pout);
	m_pnexists_given_sky_log = -logf(m_pnexists_given_sky);
	m_nopnexists_given_sky_log = -logf(1.0f-m_pnexists_given_sky);
	m_pnexists_given_ground_log = -logf(m_pnexists_given_ground);
	m_nopnexists_given_ground_log = -logf(1.0f-m_pnexists_given_ground);
	m_pnexists_given_object_log = -logf(m_pnexists_given_object);
	m_nopnexists_given_object_log = -logf(1.0f-m_pnexists_given_object);

	// Data term precomputation
	m_normalization_ground = new float[m_rows];
	m_inv_sigma2_ground = new float[m_rows];
	m_normalization_object = new float[m_max_dis];
	m_inv_sigma2_object = new float[m_max_dis];
	m_object_disparity_range = new float[m_max_dis];

	for(int i = 0; i < m_max_dis; i++) {
		float previous_mean = (float) i;
		m_object_disparity_range[i] = ComputeObjectDisparityRange(previous_mean);
	}

	// Precomputation of data term
	PrecomputeSky();
	PrecomputeObject();

	// Object Data Cost LUT
	m_obj_cost_lut = new float[m_max_dis*m_max_dis];

	for(int fn = 0; fn < m_max_dis; fn++) {
		for(int dis = 0; dis < m_max_dis; dis++) {
			m_obj_cost_lut[fn*m_max_dis+dis] = GetDataCostObject(fn, dis);
		}
	}

	const int rows_power2 = (int) powf(2, ceilf(log2f(m_rows+1)));

	CUDA_CHECK_RETURN(hipMalloc((void **)&d_disparity_big, m_rows*m_cols*sizeof(float)));
	CUDA_CHECK_RETURN(hipMalloc((void **)&d_disparity, m_rows*m_realcols*sizeof(float)));
	CUDA_CHECK_RETURN(hipMalloc((void **)&d_ground_function, m_rows*m_realcols*sizeof(float)));
	CUDA_CHECK_RETURN(hipMalloc((void **)&d_normalization_ground, m_rows*sizeof(float)));
	CUDA_CHECK_RETURN(hipMalloc((void **)&d_inv_sigma2_ground, m_rows*sizeof(float)));
	CUDA_CHECK_RETURN(hipMalloc((void **)&d_object_disparity_range, m_max_dis*sizeof(float)));
	CUDA_CHECK_RETURN(hipMalloc((void **)&d_object_lut, (rows_power2+1)*m_realcols*m_max_dis*sizeof(float)));
	CUDA_CHECK_RETURN(hipMalloc((void **)&d_obj_cost_lut, m_max_dis*m_max_dis*sizeof(float)));

	CUDA_CHECK_RETURN(hipMemcpy(d_object_disparity_range, m_object_disparity_range, sizeof(float)*m_max_dis,
			hipMemcpyHostToDevice));
	CUDA_CHECK_RETURN(hipMemcpy(d_obj_cost_lut, m_obj_cost_lut, sizeof(float)*m_max_dis*m_max_dis,
			hipMemcpyHostToDevice));

	CUDA_CHECK_RETURN(hipStreamCreate(&m_stream1));
	CUDA_CHECK_RETURN(hipStreamCreate(&m_stream2));

	m_params.rows = m_rows;
	m_params.cols = m_realcols;
	m_params.max_dis = m_max_dis;
	m_params.rows_log = m_rows_log;
	m_params.pnexists_given_sky_log = m_pnexists_given_sky_log;
	m_params.normalization_sky = m_normalization_sky;
	m_params.inv_sigma2_sky = m_inv_sigma2_sky;
	m_params.puniform_sky = m_puniform_sky;
	m_params.nopnexists_given_sky_log = m_nopnexists_given_sky_log;
	m_params.pnexists_given_ground_log = m_pnexists_given_ground_log;
	m_params.puniform = m_puniform;
	m_params.nopnexists_given_ground_log = m_nopnexists_given_ground_log;
	m_params.pnexists_given_object_log = m_pnexists_given_object_log;
	m_params.nopnexists_given_object_log = m_nopnexists_given_object_log;
	m_params.baseline = m_baseline;
	m_params.focal = m_focal;
	m_params.range_objects_z = m_range_objects_z;
	m_params.pord = m_pord;
	m_params.epsilon = m_epsilon;
	m_params.pgrav = m_pgrav;
	m_params.pblg = m_pblg;
	m_params.rows_power2 = rows_power2;
	m_params.max_sections = m_max_sections;
	m_params.max_dis_log = m_max_dis_log;
	m_params.width_margin = m_width_margin;
}

void Stixels::Finish() {
	delete[] m_cost_table;
	delete[] m_index_table;
	delete[] m_stixels;
	if(m_column_step > 1) {
		CUDA_CHECK_RETURN(hipHostFree(m_disp_im_modified));
	}
	delete[] m_column;
	delete[] m_sum;
	delete[] m_valid;
	delete[] m_ground_function;
	delete[] m_normalization_ground;
	delete[] m_inv_sigma2_ground;
	delete[] m_normalization_object;
	delete[] m_inv_sigma2_object;
	delete[] m_object_disparity_range;
	delete[] m_obj_cost_lut;
	delete[] m_log_lut;

	CUDA_CHECK_RETURN(hipFree(d_disparity_big));
	CUDA_CHECK_RETURN(hipFree(d_disparity));
	CUDA_CHECK_RETURN(hipFree(d_ground_function));
	CUDA_CHECK_RETURN(hipFree(d_normalization_ground));
	CUDA_CHECK_RETURN(hipFree(d_inv_sigma2_ground));
	CUDA_CHECK_RETURN(hipFree(d_object_disparity_range));
	CUDA_CHECK_RETURN(hipFree(d_object_lut));
	CUDA_CHECK_RETURN(hipFree(d_stixels));
	CUDA_CHECK_RETURN(hipFree(d_obj_cost_lut));

	CUDA_CHECK_RETURN(hipStreamDestroy(m_stream1));
	CUDA_CHECK_RETURN(hipStreamDestroy(m_stream2));
}



//////////////////////////////////////////////////////////////////////
// Functions
//////////////////////////////////////////////////////////////////////

void Stixels::SetDisparityImage(pixel_t *disp_im) {
	m_disp_im = disp_im;

	// New image joining the columns
	CUDA_CHECK_RETURN(hipMemcpyAsync(d_disparity_big, m_disp_im, sizeof(pixel_t)*m_rows*m_cols,
			hipMemcpyHostToDevice, m_stream1));
}

void Stixels::SetProbabilities(float pout, float pout_sky, float pground_given_nexist,
		float pobject_given_nexist, float psky_given_nexist, float pnexist_dis, float pground,
		float pobject, float psky, float pord, float pgrav, float pblg) {
	m_pout = pout;
	m_pout_sky = pout_sky;
	m_pnexists_given_ground = (pground_given_nexist*pnexist_dis)/pground;
	m_pnexists_given_object = (pobject_given_nexist*pnexist_dis)/pobject;
	m_pnexists_given_sky = (psky_given_nexist*pnexist_dis)/psky;
	m_pord = pord;
	m_pgrav = pgrav;
	m_pblg = pblg;
}

void Stixels::SetCameraParameters(int vhor, float focal, float baseline, float camera_tilt,
		float sigma_camera_tilt, float camera_height, float sigma_camera_height, float alpha_ground) {
	m_vhor = m_rows-vhor-1;
	m_focal = focal;
	m_baseline = baseline;
	m_camera_tilt = camera_tilt;
	// Degrees to radians
	m_sigma_camera_tilt = sigma_camera_tilt*(PIFLOAT)/180.0f;
	m_camera_height = camera_height;
	m_sigma_camera_height = sigma_camera_height;
	m_alpha_ground = alpha_ground;
}


void Stixels::SetDisparityParameters(const int rows, const int cols, const int max_dis,
		const float sigma_disparity_object, const float sigma_disparity_ground, float sigma_sky) {
	m_rows = rows;
	m_cols = cols;
	m_max_dis = max_dis;
	m_max_disf = (float) m_max_dis;
    	m_sigma_disparity_object = sigma_disparity_object;
    	m_sigma_disparity_ground = sigma_disparity_ground;
	m_sigma_sky = sigma_sky;
}

void Stixels::SetModelParameters(const int column_step, const bool median_step, float epsilon,
		float range_objects_z, int width_margin) {
	m_column_step = column_step;
	m_median_step = median_step;
	m_epsilon = epsilon;
	m_range_objects_z = range_objects_z;
	m_width_margin = width_margin;
}


float Stixels::Compute() {
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	// Precomputation of data term
	PrecomputeGround();

	CUDA_CHECK_RETURN(hipMemcpy(d_ground_function, m_ground_function, sizeof(float)*m_rows,
			hipMemcpyHostToDevice));
	CUDA_CHECK_RETURN(hipMemcpy(d_normalization_ground, m_normalization_ground, sizeof(float)*m_rows,
			hipMemcpyHostToDevice));
	CUDA_CHECK_RETURN(hipMemcpy(d_inv_sigma2_ground, m_inv_sigma2_ground, sizeof(float)*m_rows,
			hipMemcpyHostToDevice));

	JoinColumns<<<divUp(m_rows*m_realcols, 256), 256>>>(d_disparity_big, d_disparity, m_column_step,
			m_median_step, m_width_margin, m_rows, m_cols, m_realcols);

	hipError_t err = hipGetLastError();
	if (err != hipSuccess) {
	    printf("Error: %s %d\n", hipGetErrorString(err), err);
	}

	m_params.vhor = m_vhor;

	ComputeObjectLUT<<<m_realcols, 512>>>(d_disparity, d_obj_cost_lut, d_object_lut, m_params,
			(int) powf(2, ceilf(log2f(m_rows))));

	err = hipGetLastError();
	if (err != hipSuccess) {
	    printf("Error: %s %d\n", hipGetErrorString(err), err);
	}

	int shared_mem_size = sizeof(float)*(m_params.rows_power2*6+m_params.max_dis)
			+sizeof(int16_t)*(m_params.rows_power2*3)+
			+sizeof(pixel_t)*(m_params.rows_power2);
#if ROBUST_MEAN_ESTIMATION
	shared_mem_size += sizeof(pixel_t)*(m_params.rows_power2*2);
#endif

	StixelsKernel<<<m_realcols, m_rows,
			shared_mem_size>>>(d_disparity, m_params, d_ground_function, d_normalization_ground, d_inv_sigma2_ground,
					d_object_disparity_range, d_object_lut, d_stixels);

	err = hipGetLastError();
	if (err != hipSuccess) {
	    printf("Error: %s %d\n", hipGetErrorString(err), err);
	}

	// Synchronize
	CUDA_CHECK_RETURN(hipDeviceSynchronize());

	hipEventRecord(stop, 0);
	float elapsed_time_ms;
	hipEventElapsedTime(&elapsed_time_ms, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	CUDA_CHECK_RETURN(hipMemcpy(m_stixels, d_stixels, m_realcols*m_max_sections*sizeof(Section),
			hipMemcpyDeviceToHost));
	return elapsed_time_ms;
}

Section* Stixels::GetStixels() {
	return m_stixels;
}

int Stixels::GetRealCols() {
	return m_realcols;
}

int Stixels::GetMaxSections() {
	return m_max_sections;
}

float Stixels::FastLog(float v) {
	return m_log_lut[(int)((v)*LOG_LUT_SIZE+0.5f)];
}

pixel_t Stixels::ComputeMean(const int vB, const int vT, const int u) {
	const pixel_t valid_dif = m_valid[u*(m_rows+1)+vT+1]-m_valid[u*(m_rows+1)+vB];
	pixel_t mean = (valid_dif == 0) ? 0 : (m_sum[u*(m_rows+1)+vT+1]-m_sum[u*(m_rows+1)+vB])/valid_dif;

#if ROBUST_MEAN_ESTIMATION
		pixel_t total_weight = 0;
		pixel_t total_dv = 0;
		for(int v = vB; v <= vT; v++) {
			const pixel_t dv = m_column[u*m_rows+v];
			const int val = dv != INVALID_DISPARITY;
			const pixel_t weight = (pixel_t)val/(1+fabsf(dv-mean));
			total_weight += weight;
			total_dv += weight*dv;
		}
		mean = (total_weight == 0.0f) ? 0.0f : total_dv/total_weight;
#endif

	return mean;
}

void Stixels::PrecomputeGround() {
	const float fb = (m_focal*m_baseline)/m_camera_height;
	const float pout = m_pout;

	for(int v = 0; v < m_rows; v++) {
		const float fn = GroundFunction(v);
		m_ground_function[v] = fn;

		const float x = m_camera_tilt+(float)(m_vhor-v)/m_focal;
		const float sigma2_road = fb*fb*(m_sigma_camera_height*m_sigma_camera_height
				*x*x/(m_camera_height*m_camera_height)+m_sigma_camera_tilt*m_sigma_camera_tilt);
        const float sigma = sqrtf(m_sigma_disparity_ground*m_sigma_disparity_ground+sigma2_road);

		const float a_range = 0.5f*(erf((m_max_disf-fn)/(sigma*sqrtf(2.0f)))-erf((-fn)/(sigma*sqrtf(2.0f))));

		m_normalization_ground[v] = FastLog(a_range) - FastLog((1.0f - pout)/(sigma*sqrtf(2.0f*PIFLOAT)));
		m_inv_sigma2_ground[v] = 1.0f/(2.0f*sigma*sigma);
	}
}

void Stixels::PrecomputeObject() {
	const float pout = m_pout;

	for(int dis = 0; dis < m_max_dis; dis++) {
		const float fn = (float) dis;

		const float sigma_object = fn*fn*m_range_objects_z/(m_focal*m_baseline);
        const float sigma = sqrtf(m_sigma_disparity_object*m_sigma_disparity_object+sigma_object*sigma_object);

		const float a_range = 0.5f*(erf((m_max_disf-fn)/(sigma*sqrtf(2.0f)))-erf((-fn)/(sigma*sqrtf(2.0f))));

		m_normalization_object[dis] = FastLog(a_range) - FastLog((1.0f - pout)/(sigma*sqrtf(2.0f*PIFLOAT)));
		m_inv_sigma2_object[dis] = 1.0f/(2.0f*sigma*sigma);
	}
}

float Stixels::GetDataCostObject(const int fn, const int dis){
	float data_cost = m_pnexists_given_object_log;
	if(!ALLOW_INVALID_DISPARITIES || dis != INVALID_DISPARITY) {
		const float model_diff = (float) (dis-fn);
		const float pgaussian = m_normalization_object[fn] + model_diff*model_diff*m_inv_sigma2_object[fn];

		const float p_data = fminf(m_puniform, pgaussian);
		data_cost = p_data + m_nopnexists_given_object_log;
	}
	return data_cost;
}

void Stixels::PrecomputeSky() {
	const float sigma = m_sigma_sky;
	const float pout = m_pout_sky;

	const float a_range = 0.5f*(erf(m_max_disf/(sigma*sqrtf(2.0f)))-erf(0.0f));
	m_normalization_sky = FastLog(a_range) - logf((1.0f - pout)/(sigma*sqrtf(2.0f*PIFLOAT)));
	m_inv_sigma2_sky = 1.0f/(2.0f*sigma*sigma);
}

float Stixels::GroundFunction(const int v) {
	return m_alpha_ground*(float)(m_vhor-v);
}

float Stixels::ComputeObjectDisparityRange(const float previous_mean) {
	float range_disp = 0.0f;
	if(previous_mean != 0) {
		const float pmean_plus_z = (m_baseline*m_focal/previous_mean) + m_range_objects_z;
		range_disp = previous_mean - (m_baseline*m_focal/pmean_plus_z);
	}
	return range_disp;
}
